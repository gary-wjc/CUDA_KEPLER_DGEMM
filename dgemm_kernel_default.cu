//nvcc -arch=sm_xx --shared -DNDEBUG --compiler-options -fPIC,-O3 -Xcicc -O3 -keep -Xptxas -v uuu.cu -o libdgemm_kernel.so
#if __CUDA_ARCH__ >= 800
#define DGEMM_SM80
#endif


#include <hip/hip_runtime.h>
#include <cassert>
#include <cstddef>

struct GLoadFirstPos {
  const unsigned m_mMod16, m_kMod16;
  __device__ GLoadFirstPos(unsigned warp_id, bool row_major):
    m_mMod16(row_major ? ((threadIdx.x & 1) | (warp_id << 1)) : (threadIdx.x & 15)),
    m_kMod16(row_major ? ((threadIdx.x & 14) | (threadIdx.x >> 4)) :
      ((warp_id << 1) | (threadIdx.x >> 4))) {

    assert(blockDim.x == 32 && warp_id < 8);
  }
};

class MemTransfer {
  unsigned m_sharedFirstPos;
  const char *m_gPtr;
  std::size_t m_kStrideBytes;
  std::size_t m_mnStrideBytes;
  unsigned m_mnTasks;
  std::size_t m_kTasks;
public:
  __device__ MemTransfer(const double *A, const double *B,
    std::size_t M, std::size_t N, std::size_t K, std::size_t LDA,
    std::size_t LDB, bool a_rowmajor, bool b_rowmajor,
    std::size_t block_start_m, std::size_t block_start_n) {
    assert(blockDim.y * blockDim.z == 16);
    const int warp_id = threadIdx.z * blockDim.y + threadIdx.y;
    const bool select_b = warp_id & 8;
    const bool kmajor = select_b ? !b_rowmajor : a_rowmajor;
    const GLoadFirstPos first_pos(warp_id & 7, kmajor);
    const unsigned &koff = first_pos.m_kMod16;
    const unsigned &mnoff = first_pos.m_mMod16;
    m_sharedFirstPos = koff == 15 ? ((mnoff >> 1) * 34u | (mnoff & 1)) :
      ((koff >> 1) * 34u + 2u + (mnoff | ((koff & 1) << 4)));
    std::size_t mn_max = select_b ? N : M;
    std::size_t mn_first = (select_b ? block_start_n : block_start_m) + mnoff;
    std::size_t LD = select_b ? LDB : LDA;
    m_kStrideBytes = (kmajor ? 1ul : LD) << 7;
    m_mnStrideBytes = (kmajor ? LD : 1ul) << 7;
    m_gPtr = reinterpret_cast<const char*>(select_b ? B : A) +
      (m_mnStrideBytes >> 4) * mn_first + (m_kStrideBytes >> 4) * koff;
    auto get_tasks = [](std::size_t start, std::size_t end)->std::size_t {
      if (end <= start) return 0;
      return (end + 15u - start) >> 4;
    };
    std::size_t mn_tasks = get_tasks(mn_first, mn_max);
    m_mnTasks = mn_tasks > 8 ? 8u : mn_tasks;
    m_kTasks = get_tasks(koff, K);
  }
#ifdef DGEMM_SM80
#pragma message "enable cp.async"
  __device__ void issue_transfer(double2 *shared) {
    unsigned tbytes = 0;
    if (m_kTasks) {
      tbytes = 8;
      m_kTasks--;
    }
    auto gptr = m_gPtr;
    asm ("cvta.to.global.u64 %0,%0;":"+l"(gptr));
    unsigned baseoff = (threadIdx.z * blockDim.y + threadIdx.y & 8) << 8;
    double *begin = (&shared[0].x) + baseoff + m_sharedFirstPos;
    asm ("cvta.to.shared.u64 %0,%0;":"+l"(begin));
#pragma unroll
    for (int j = 0; j < 8; ++j) {
      if (j >= m_mnTasks) break;
      asm volatile ("cp.async.ca.shared.global [%0],[%1],8,%2;"
        ::"l"(begin),"l"(gptr),"r"(tbytes));
      gptr += m_mnStrideBytes;
      begin += 256;
    }
    m_gPtr += m_kStrideBytes;
    asm volatile ("cp.async.commit_group;");
  }
#endif
  __device__ void load(double (&recv)[8]) {
    if (m_kTasks) {
      m_kTasks--;
      auto gptr = m_gPtr;
#pragma unroll
      for (int j = 0; j < 8; ++j) {
        if (j >= m_mnTasks) break;
        recv[j] = __ldg((const double*)gptr);
        gptr += m_mnStrideBytes;
      }
      m_gPtr += m_kStrideBytes;
    } else {
#pragma unroll
      for (int j = 0; j < 8; ++j) {
        if (j >= m_mnTasks) break;
        recv[j] = 0;
      }
    }
  }
  __device__ void store(const double (&recv)[8], double2 *shared) const {
    unsigned baseoff = (threadIdx.z * blockDim.y + threadIdx.y & 8) << 8;
    double *begin = (&shared[0].x) + baseoff + m_sharedFirstPos;
#pragma unroll
    for (int j = 0; j < 8; ++j) {
      if (j < m_mnTasks) begin[j << 8] = recv[j];
    }
  }
};

class WarpAccumulator {
  double2 m_acc[2][2][2][2];
  bool m_calcValid;
public:
  __device__ WarpAccumulator(std::size_t M, std::size_t N,
    std::size_t block_m_base, std::size_t block_n_base): m_acc{} {

    assert(blockDim.y == 4 && blockDim.z == 4 && blockDim.x == 32);
    std::size_t npos_start = block_n_base + (threadIdx.y << 5);
    std::size_t mpos_start = block_m_base + (threadIdx.z << 5);
    m_calcValid = npos_start < N && mpos_start < M;
  }
  __device__ void acc_k16(const double2 *shared) {
    if (!m_calcValid) return;
#ifdef DGEMM_SM80
#pragma message "use tensor core"
    unsigned mnstartdiv2 = threadIdx.x >> 2;
    unsigned kstartdiv2 = (threadIdx.x & 3) << 1;
    unsigned offins32 = mnstartdiv2 + 1 + kstartdiv2 * 17u;
    const double2 *abase = &shared[threadIdx.z * 256u];
    const double2 *bbase = &shared[(threadIdx.y+4) * 256u];
    const double2 *astart = &abase[offins32];
    const double2 *bstart = &bbase[offins32];
    auto acc_k4 = [&]()->void {
      double2 a[2] = { *astart, astart[128] };
      double2 b[2] = { *bstart, bstart[128] };
#pragma unroll
      for (short nseg = 0; nseg < 2; ++nseg) {
        const double2 &bv = b[nseg];
#pragma unroll
        for (short mseg = 0; mseg < 2; ++mseg) {
          const double2 &av = a[mseg];
          asm (
#if __CUDA_ARCH__ >= 900
#pragma message "optimize for Hopper arch"
            "mma.sync.aligned.m16n8k4.row.col.f64.f64.f64.f64"
	      " {%0,%1,%4,%5},{%8,%9},{%10},{%0,%1,%4,%5};"
            "mma.sync.aligned.m16n8k4.row.col.f64.f64.f64.f64"
	      " {%2,%3,%6,%7},{%8,%9},{%11},{%2,%3,%6,%7};"
#else
            "mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 {%0,%1},{%8},{%10},{%0,%1};"
            "mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 {%2,%3},{%8},{%11},{%2,%3};"
            "mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 {%4,%5},{%9},{%10},{%4,%5};"
            "mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 {%6,%7},{%9},{%11},{%6,%7};"
#endif
           :"+d"(m_acc[nseg][mseg][0][0].x),"+d"(m_acc[nseg][mseg][0][1].x),
            "+d"(m_acc[nseg][mseg][0][0].y),"+d"(m_acc[nseg][mseg][0][1].y),
            "+d"(m_acc[nseg][mseg][1][0].x),"+d"(m_acc[nseg][mseg][1][1].x),
            "+d"(m_acc[nseg][mseg][1][0].y),"+d"(m_acc[nseg][mseg][1][1].y)
           :"d"(bv.x),"d"(bv.y),"d"(av.x),"d"(av.y));
        }
      }
    };
    acc_k4();
    astart += 8u;
    bstart += 8u;
    acc_k4();
    astart += 9u;
    bstart += 9u;
    acc_k4();
    if (kstartdiv2 == 6) offins32 = mnstartdiv2 * 17u;
    else offins32 += 25u;
    astart = &abase[offins32];
    bstart = &bbase[offins32];
    acc_k4();
#else
    unsigned mstartdiv2 = (threadIdx.x & 3) << 1;
    unsigned nstartdiv2 = threadIdx.x >> 2;
    const double2 *astart = &shared[threadIdx.z * 256u + mstartdiv2 + 1];
    const double2 *bstart = &shared[(threadIdx.y+4) * 256u + nstartdiv2 + 1];

    auto acc_k1 = [&](unsigned short a2sep)->void {
      double2 a[2][2];
      a[0][0] = *astart;
      a[0][1] = astart[a2sep];
      a[1][0] = astart[128];
      a[1][1] = astart[128+a2sep];
      double2 b[2];
      b[0] = *bstart;
      b[1] = bstart[128];
#pragma unroll
      for (short nseg = 0; nseg < 2; ++nseg) {
        const double2 &b1 = b[nseg];
#pragma unroll
	for (short mseg = 0; mseg < 2; ++mseg) {
          const double2 &a1 = a[mseg][0];
          const double2 &a2 = a[mseg][1];

          m_acc[nseg][mseg][0][0].x += a1.x * b1.x; //m0
          m_acc[nseg][mseg][0][0].y += a1.y * b1.x; //m1
          m_acc[nseg][mseg][0][1].x += a2.x * b1.x; //m2
          m_acc[nseg][mseg][0][1].y += a2.y * b1.x; //m3

          m_acc[nseg][mseg][1][0].x += a1.x * b1.y;
          m_acc[nseg][mseg][1][0].y += a1.y * b1.y;
          m_acc[nseg][mseg][1][1].x += a2.x * b1.y;
          m_acc[nseg][mseg][1][1].y += a2.y * b1.y;
        }
      }
    };

#pragma unroll
    for (int kd2 = 0; kd2 < 7; ++kd2) {
      acc_k1(1);
      astart += 8;
      bstart += 8;
      acc_k1(1);
      astart += 9;
      bstart += 9;
    }
    acc_k1(1);
    astart = &shared[threadIdx.z * 256u + mstartdiv2 * 17];
    bstart = &shared[(threadIdx.y+4) * 256u + nstartdiv2 * 17];
    acc_k1(17);
#endif
  }
  __device__ void store(double *C, std::size_t M, std::size_t N, std::size_t LDC,
    std::size_t block_m_base, std::size_t block_n_base) {

    if (!m_calcValid) return;
    std::size_t npos_start = block_n_base + (threadIdx.y << 5) +
      (threadIdx.x & 1) + (threadIdx.x >> 2 << 1);
    std::size_t mpos_start = block_m_base + (threadIdx.z << 5) +
      ((threadIdx.x & 2) << 2);
    #pragma unroll
    for (short nseg = 0; nseg < 2; ++nseg) {
      #pragma unroll
      for (short mseg = 0; mseg < 2; ++mseg) {
        auto shfl = [](double2 var, int idx)->double2 {
          return make_double2(__shfl_sync(0xFFFFFFFF, var.x, idx),
            __shfl_sync(0xFFFFFFFF, var.y, idx));
        };
        m_acc[nseg][mseg][1][0] = shfl(m_acc[nseg][mseg][1][0], threadIdx.x ^ 1);
        m_acc[nseg][mseg][1][1] = shfl(m_acc[nseg][mseg][1][1], threadIdx.x ^ 1);
        if (threadIdx.x & 1) {
          double2 t0 = m_acc[nseg][mseg][1][0];
          double2 t1 = m_acc[nseg][mseg][1][1];
          m_acc[nseg][mseg][1][0] = m_acc[nseg][mseg][0][0];
          m_acc[nseg][mseg][1][1] = m_acc[nseg][mseg][0][1];
          m_acc[nseg][mseg][0][0] = t0;
          m_acc[nseg][mseg][0][1] = t1;
        }
        std::size_t mpos = mpos_start + mseg * 16u;
        std::size_t npos = npos_start + nseg * 16u;
        auto store2 = [&](double2 val)->void {
          if (npos < N) {
            if (mpos < M) C[mpos * LDC + npos] += val.x;
            if (mpos + 1u < M) C[(mpos+1u) * LDC + npos] += val.y;
          }
        };
	store2(m_acc[nseg][mseg][0][0]);
        mpos += 2;
        store2(m_acc[nseg][mseg][0][1]);
        mpos += 2;
        npos ^= 1;
        store2(m_acc[nseg][mseg][1][0]);
        mpos += 2;
        store2(m_acc[nseg][mseg][1][1]);
      }
    }
  }
};

__launch_bounds__(512, 1) __global__ void dgemm_kernel(
  const double *A, const double *B, double *C,
  std::size_t M, std::size_t N, std::size_t K, bool a_rowmajor, bool b_rowmajor,
  std::size_t LDA, std::size_t LDB, std::size_t LDC) {

  assert(blockDim.y == 4 && blockDim.z == 4);
  unsigned nblkid = blockIdx.y;
  unsigned mblkid = blockIdx.z;
  const unsigned nblks = (N+127u) >> 7;
  const unsigned mblks = (M+127u) >> 7;
  if (nblkid < (nblks & unsigned(-4)) &&
    mblkid < (mblks & unsigned(-4))) {
    //perform reordered mapping of (4, nblks&-4) to increase L2-cache hit rate
    unsigned short submid = nblkid & 3u;
    if (nblkid & 4u) submid = 3u - submid;
    unsigned new_mblkid = (mblkid & unsigned(-4)) | submid;
    nblkid = (nblkid >> 2) + (nblks >> 2) * (mblkid & 3u);
    mblkid = new_mblkid;
  }
  const std::size_t block_m_base = std::size_t(mblkid) << 7;
  const std::size_t block_n_base = std::size_t(nblkid) << 7;
  if (block_m_base >= M || block_n_base >= N) return;
  MemTransfer mtr(A, B, M, N, K, LDA, LDB, a_rowmajor, b_rowmajor,
    block_m_base, block_n_base);
  WarpAccumulator acc(M, N, block_m_base, block_n_base);

  extern __shared__ double2 shared[]; //double2 shared[2048];
#ifdef DGEMM_SM80
#pragma message "use double-buffered shared memory"
  __shared__ double2 shared2[2048];
  mtr.issue_transfer(shared);
  for (std::size_t k = 0; k < K; k += 16) {
    mtr.issue_transfer(k & 16u ? shared : shared2);
    asm volatile ("cp.async.wait_group 1;");
    __syncthreads();
    acc.acc_k16(k & 16u ? shared2 : shared);
  }
#else
  double gload[8];
  for (std::size_t k = 0; k < K; k += 16) {
    mtr.load(gload);
    __syncthreads();
    mtr.store(gload, shared);
    __syncthreads();
    acc.acc_k16(shared);
  }
#endif
  acc.store(C, M, N, LDC, block_m_base, block_n_base);
}

void dgemm_async(hipStream_t &stream, const double *devA, const double *devB, double *devC,
  std::size_t M, std::size_t N, std::size_t K, bool a_rowmajor, bool b_rowmajor,
  std::size_t LDA, std::size_t LDB, std::size_t LDC) noexcept {

  unsigned shared_bytes = 32768;
  hipFuncSetAttribute(reinterpret_cast<const void*>(&dgemm_kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
    shared_bytes);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(&dgemm_kernel), hipSharedMemBankSizeEightByte);

  //TODO: when gridDim.x or gridDim.y exceed device capability, call dgemm_kernel in loops
  const dim3 grid_size(1, (N+127u) >> 7, (M+127u) >> 7);
  const dim3 block_size(32, 4, 4);

  dgemm_kernel<<<grid_size, block_size, shared_bytes, stream>>>(
    devA, devB, devC, M, N, K,
    a_rowmajor, b_rowmajor, LDA, LDB, LDC);
}
